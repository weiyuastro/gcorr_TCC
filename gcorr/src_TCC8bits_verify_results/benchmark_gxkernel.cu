#include "hip/hip_runtime.h"
// TCC related parameters 
#define NR_BITS 8
//#define NR_RECEIVERS_PER_BLOCK 32
//#define NR_TIMES_PER_BLOCK (1024 / (NR_BITS))
#define transpose_M 32
#define transpose_N 32

#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>
#include <strings.h>
#include <argp.h>
#include <complex.h>
#include <hip/hip_complex.h>
#include <npp.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipfft/hipfft.h>
#include "gxkernel.h"


// tensor core correlator
#include <iostream>
#include "test/Common/Record.h"
#include "util/ExceptionPropagator.h"
#include "util/hip/hiprtc.h"
#include "test/Common/ComplexInt4.h"
#include "test/Common/UnitTest.h"
//#include "test/CorrelatorTest/Options.h"
#include "libtcc/Correlator.h"
#include "util/multi_array.h"
#include "util/cu.h"
#include <hip/hip_fp16.h>
#include <hiprand/hiprand_kernel.h>


#if NR_BITS == 4
typedef complex_int4_t	      Sample;
typedef std::complex<int32_t> Visibility;
#elif NR_BITS == 8
typedef std::complex<int8_t>  Sample;
typedef std::complex<int32_t> Visibility;
#elif NR_BITS == 16
typedef std::complex<__half>  Sample;
typedef std::complex<float>   Visibility;
#endif



__global__ void generateRandomInt8(int8_t* random_values, int num_values) {
    hiprandState state;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(1234, idx, 0, &state);
    float random_float = hiprand_uniform(&state);
    int8_t random_int8 = static_cast<int8_t>(random_float * 256 - 128);
    if (idx < num_values) {
        random_values[idx] = random_int8;
    }
}


__global__ void GPU_Transpose_8bits(const hipfftComplex *  PFB_FFT_out_Pol0, char2 * Transpose_out_data, int nr_channels, int samples_per_channel)
{

	int res_x;
	int res_y;

  __shared__ float2 tmp0[transpose_M][transpose_N];

  int in_x = blockIdx.x * blockDim.x + threadIdx.x;
  int in_y = blockDim.y * gridDim.y * blockIdx.z + blockIdx.y * blockDim.y + threadIdx.y;

	tmp0[threadIdx.y][threadIdx.x] = PFB_FFT_out_Pol0[in_x + in_y * blockDim.x * gridDim.x];

  __syncthreads();

  int new_x = threadIdx.y;
  int new_y = threadIdx.x;

  int to_x = blockIdx.y * blockDim.y + threadIdx.x;
  int to_y = blockIdx.x * blockDim.x + threadIdx.y;
  int to_z = blockIdx.z;

	// Convert to s8, round to nearest integer, and saturate
	asm("cvt.rni.sat.s8.f32 %0, %1;" : "=r"(res_x) : "f"(tmp0[new_y][new_x].x));
	asm("cvt.rni.sat.s8.f32 %0, %1;" : "=r"(res_y) : "f"(tmp0[new_y][new_x].y));

  Transpose_out_data[to_z * nr_channels * samples_per_channel + to_y * samples_per_channel + to_x] = make_char2((char) res_x, (char) res_y);
}


__global__ void GPU_Reorder_after_transpose_8bits(const char2 *  Transpose_out_data, char2 * GPU_out_data)
{

  int idx_in, idx_out;

  idx_in =  threadIdx.x + blockDim.x * blockIdx.x + blockIdx.y * (blockDim.x * gridDim.x) + threadIdx.y * (blockDim.x * gridDim.x * gridDim.y) + blockIdx.z * (blockDim.x * blockDim.y * gridDim.x * gridDim.y);
  idx_out = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * blockIdx.z + blockDim.x * blockDim.y * gridDim.z * blockIdx.x + blockDim.x * blockDim.y * gridDim.z * gridDim.x * blockIdx.y;

  GPU_out_data[idx_out] = Transpose_out_data[idx_in];
}


__global__ void GPU_set_initial_value(char2 * samples, int dim1, int dim2, int dim3, int dim4, int dim5, int dim3_cpy)
{


  int idx_out;
  idx_out = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * blockIdx.z + blockDim.x * blockDim.y * gridDim.z * blockIdx.x + blockDim.x * blockDim.y * gridDim.z * gridDim.x * blockIdx.y;


  if((threadIdx.x==dim1) && (threadIdx.y==dim2) && (blockIdx.z==dim3) && (blockIdx.x==dim4) && (blockIdx.y==dim5))
  {
    samples[idx_out].x = 2;
    samples[idx_out].y = 3;
  }
  else if((threadIdx.x==dim1) && (threadIdx.y==dim2) && (blockIdx.z==dim3_cpy) && (blockIdx.x==dim4) && (blockIdx.y==dim5))
  {
    samples[idx_out].x = 4;
    samples[idx_out].y = 5;
  }
}


__global__ void complex2char(const hipComplex *  PFB_FFT_out_Pol0, char2 * GPU_out_data)
{

	int res_x = 0;
	int res_y = 0;

  unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  unsigned int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
  unsigned int thread_idx = ((gridDim.x * blockDim.x) * idy) + idx;

	// Convert to s8, round to nearest integer, and saturate
	asm("cvt.rni.sat.s8.f32 %0, %1;" : "=r"(res_x) : "f"(PFB_FFT_out_Pol0[thread_idx].x));
	asm("cvt.rni.sat.s8.f32 %0, %1;" : "=r"(res_y) : "f"(PFB_FFT_out_Pol0[thread_idx].y));

  GPU_out_data[thread_idx] = make_char2((char) res_x, (char) res_y);
}



/*
 * Code to test the kernels in the gxkernel.cu.
 */

void preLaunchCheck() {
  hipError_t error;

  error = hipGetLastError();
  
  if (error != hipSuccess) {
    fprintf(stderr, "Error: Previous CUDA failure: \"%s\". Exiting\n",
	    hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

void postLaunchCheck() {
  hipError_t error;

  error = hipGetLastError();
  
  if (error != hipSuccess) {
    fprintf(stderr, "Error: Failure Launching kernel: \"%s\". Exiting\n",
	    hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

struct timerCollection {
  hipEvent_t startTime;
  hipEvent_t endTime;
  int nTimers;
  char **timerNames;
  int *numIterations;
  float **timerResults;
  float **timerStatistics;
  int *timerCalculated;
  int currentTimer;
};

void timerInitialise(struct timerCollection *tc) {
  // Set up the structure correctly
  gpuErrchk(hipEventCreate(&(tc->startTime)));
  gpuErrchk(hipEventCreate(&(tc->endTime)));
  tc->nTimers = 0;
  tc->timerNames = NULL;
  tc->numIterations = NULL;
  tc->timerResults = NULL;
  tc->timerStatistics = NULL;
  tc->timerCalculated = NULL;
  tc->currentTimer = -1;
}

void timerAdd(struct timerCollection *tc, const char* timerName) {
  // Add a timer to the collector.
  tc->nTimers ++;
  tc->timerNames = (char **)realloc(tc->timerNames, tc->nTimers * sizeof(char *));
  tc->timerNames[tc->nTimers - 1] = (char *)malloc(256 * sizeof(char));
  strcpy(tc->timerNames[tc->nTimers - 1], timerName);
  tc->numIterations = (int *)realloc(tc->numIterations, tc->nTimers * sizeof(int));
  tc->numIterations[tc->nTimers - 1] = 0;
  tc->timerResults = (float **)realloc(tc->timerResults, tc->nTimers * sizeof(float *));
  tc->timerResults[tc->nTimers - 1] = NULL;
  tc->timerStatistics = (float **)realloc(tc->timerStatistics, tc->nTimers * sizeof(float *));
  tc->timerCalculated = (int *)realloc(tc->timerCalculated, tc->nTimers * sizeof(int));
  tc->timerCalculated[tc->nTimers - 1] = 0;
}

int timerStart(struct timerCollection *tc, const char *timerName) {
  // Start the timer.
  // Return immediately if a timer has already been started.
  if (tc->currentTimer != -1) {
    printf("timerStart: another timer is already running!\n");
    return -1;
  }
  
  int i;
  for (i = 0; i < tc->nTimers; i++) {
    if (strcmp(tc->timerNames[i], timerName) == 0) {
      tc->currentTimer = i;
      break;
    }
  }

  if (tc->currentTimer >= 0) {
    tc->timerCalculated[tc->currentTimer] = 0;
    gpuErrchk(hipEventRecord(tc->startTime, 0));
    return 0;
  }

  printf("timerStart: could not find timer entry for %s!\n", timerName);
  return -2;
}

float timerEnd(struct timerCollection *tc) {

  // Catch any kernel-launch errors
  gpuErrchk(hipPeekAtLastError());

  // Stop the running timer.
  // Return immediately if no timer has been started.
  if (tc->currentTimer == -1) {
    return -1.0f;
  }

  // Keep a copy of the current timer.
  int ct = tc->currentTimer;
  
  // Stop the timer.
  float elapsed_ms = -1.0f;
  gpuErrchk(hipEventRecord(tc->endTime, 0));
  gpuErrchk(hipEventSynchronize(tc->endTime));
  gpuErrchk(hipEventElapsedTime(&elapsed_ms, tc->startTime, tc->endTime));

  // Add an iteration to the right place.
  tc->numIterations[ct] += 1;
  int nint = tc->numIterations[ct];
  tc->timerResults[ct] = (float *)realloc(tc->timerResults[ct],
					  nint * sizeof(float));
  if (tc->timerResults[ct] == NULL) {
    printf("timerEnd realloc failed\n");
    exit(0);
  }  
  tc->timerResults[ct][nint - 1] = elapsed_ms;

  // Reset the current timer.
  tc->currentTimer = -1;
  
  // Return the elapsed time.
  //return tc->timerResults[ct][nint];
  return elapsed_ms;
}

void time_stats_single(float *timearray, int ntime, float **output) {
  int i = 0;
  *output = (float *)malloc(3 * sizeof(float));

  (*output)[0] = 0.0;
  for (i = 1; i < ntime; i++) {
    (*output)[0] += timearray[i];
    if (i == 1) {
      (*output)[1] = timearray[i];
      (*output)[2] = timearray[i];
    } else {
      (*output)[1] = (timearray[i] < (*output)[1]) ? timearray[i] : (*output)[1];
      (*output)[2] = (timearray[i] > (*output)[2]) ? timearray[i] : (*output)[2];
    }
  }

  if ((ntime - 1) > 0) {
    (*output)[0] /= (float)(ntime - 1);
  }

  return;
			   
}


void prepareJson(FILE **fp, char *filename) {
  *fp = fopen(filename, "w");
  fprintf(*fp, "{ ");
  printf("JSON file %s opened for writing\n", filename);
}

void closeJson(FILE *fp) {
  if (fp == NULL) return;

  fprintf(fp, " }\n");
  fclose(fp);
}

void startJsonObject(FILE *fp, const char *tag, int first) {
  if (fp == NULL) return;

  if (first == 1) {
    fprintf(fp, "\"%s\":", tag);
  } else {
    fprintf(fp, ",\"%s\":", tag);
  }    
  fprintf(fp, "{");
}

void endJsonObject(FILE *fp) {
  if (fp == NULL) return;

  fprintf(fp, "}");
}

void writeJsonValue(FILE *fp, const char *type, int first, const char *tag, ...) {
  if (fp == NULL) return;

  va_list ap;
  va_start(ap, tag);
  
  if (first == 1) {
    fprintf(fp, "\"%s\":", tag);
  } else {
    fprintf(fp, ",\"%s\":", tag);
  }    
  if (strcmp(type, "int") == 0) {
    int v = va_arg(ap, int);
    fprintf(fp, "%d", v);
  } else if (strcmp(type, "float") == 0) {
    double v = va_arg(ap, double);
    fprintf(fp, "%f", (float)v);
  } else if (strcmp(type, "string") == 0) {
    char *v = va_arg(ap, char*);
    fprintf(fp, "\"%s\"", v);
  }
}

void writeJsonArray(FILE *fp, const char *type, int first, const char *tag, int len, ...) {
  if (fp == NULL) return;

  va_list ap;
  va_start(ap, len);
  
  if (first == 1) {
    fprintf(fp, "\"%s\": [", tag);
  } else {
    fprintf(fp, ",\"%s\": [", tag);
  }    
  int i;
  if (strcmp(type, "int") == 0) {
    int *iarr = va_arg(ap, int*);
    for (i = 0; i < len; i++) {
      if (i > 0) {
	fprintf(fp, ",");
      }
      fprintf(fp, "%d", iarr[i]);
    }
  } else if (strcmp(type, "float") == 0) {
    float *farr = va_arg(ap, float*);
    for (i = 0; i < len; i++) {
      if (i > 0) {
	fprintf(fp, ",");
      }
      fprintf(fp, "%f", farr[i]);
    }
  } else if (strcmp(type, "string") == 0) {
    char **carr = va_arg(ap, char**);
    for (i = 0; i < len; i++) {
      if (i > 0) {
	fprintf(fp, ",");
      }
      fprintf(fp, "\"%s\"", carr[i]);
    }
  }
  fprintf(fp, "]");
}

void timerPrintStatistics(struct timerCollection *tc, const char *timerName,
			  float implied_time, FILE *fp) {
  // Calculate statistics if required and print the output.
  int i, c = -1;

  // Find the appropriate timer.
  for (i = 0; i < tc->nTimers; i++) {
    if (strcmp(tc->timerNames[i], timerName) == 0) {
      c = i;
      break;
    }
  }

  if (c >= 0) {
    if (tc->timerCalculated[c] == 0) {
      // Calculate the statistics.
      (void)time_stats_single(tc->timerResults[c], tc->numIterations[c],
			      &(tc->timerStatistics[c]));
      tc->timerCalculated[c] = 1;
    }
    startJsonObject(fp, timerName, 0);
    printf("\n==== TIMER: %s ====\n", tc->timerNames[c]);
    printf("Iterations | Average time |  Min time   |  Max time   | Data time  | Speed up  |\n");
    printf("%5d      | %8.3f ms  | %8.3f ms | %8.3f ms | %8.3f s | %8.3f  |\n",
	   (tc->numIterations[c] - 1), (tc->timerStatistics[c][0]),
	   (tc->timerStatistics[c][1]), (tc->timerStatistics[c][2]),
	   implied_time, ((implied_time * 1e3) / tc->timerStatistics[c][0]));
    writeJsonValue(fp, "int", 1, "niterations", (tc->numIterations[c] - 1));
    writeJsonValue(fp, "float", 0, "average", tc->timerStatistics[c][0]);
    writeJsonValue(fp, "float", 0, "minimum", tc->timerStatistics[c][1]);
    writeJsonValue(fp, "float", 0, "maximum", tc->timerStatistics[c][2]);
    writeJsonValue(fp, "float", 0, "speedup", ((implied_time * 1e3) / tc->timerStatistics[c][0]));
    
    endJsonObject(fp);
  }
}

const char *argp_program_version = "benchmark_gxkernel 1.0";
static char doc[] = "benchmark_gxkernel -- testing performance of various kernels";
static char args_doc[] = "";

/* Our command line options */
static struct argp_option options[] = {
  { "loops", 'n', "NLOOPS", 0, "run each performance test NLOOPS times" },
  { "threads", 't', "NTHREADS", 0, "run with NTHREADS threads on each test" },
  { "antennas", 'a', "NANTENNAS", 0, "assume NANTENNAS antennas when required" },
  { "channels", 'c', "NCHANNELS", 0, "assume NCHANNELS frequency channels when required" },
  { "samples", 's', "NSAMPLES", 0, "assume NSAMPLES when unpacking" },
  { "bandwidth", 'b', "BANDWIDTH", 0, "the bandwidth in Hz" },
  { "verbose", 'v', 0, 0, "output more" },
  { "bits", 'B', "NBITS", 0, "number of bits assumed in the data" },
  { "complex", 'I', 0, 0, "the data input is complex sampled" },
  { "json", 'j', "JSONFILE", 0, "output the timing data to this JSON file" },
  { 0 }
};

struct arguments {
  int nloops;
  int nthreads;
  int nantennas;
  int nchannels;
  int nsamples;
  int bandwidth;
  int verbose;
  int nbits;
  int complexdata;
  char jsonfile[256];
};

/* The option parser */
static error_t parse_opt(int key, char *arg, struct argp_state *state) {
  struct arguments *arguments = (struct arguments *)state->input;

  switch (key) {
  case 'n':
    arguments->nloops = atoi(arg);
    break;
  case 't':
    arguments->nthreads = atoi(arg);
    break;
  case 'a':
    arguments->nantennas = atoi(arg);
    break;
  case 'c':
    arguments->nchannels = atoi(arg);
    break;
  case 's':
    arguments->nsamples = atoi(arg);
    break;
  case 'b':
    arguments->bandwidth = atoi(arg);
    break;
  case 'v':
    arguments->verbose = 1;
    break;
  case 'B':
    arguments->nbits = atoi(arg);
    break;
  case 'C':
    arguments->complexdata = 1;
    break;
  case 'j':
    strncpy(arguments->jsonfile, arg, 256);
    break;
  }
  return 0;
}

/* The argp parser */
static struct argp argp = { options, parse_opt, args_doc, doc };



inline void checkCudaCall(hipError_t error)
{
  if (error != hipSuccess) {
    std::cerr << "error " << error << std::endl;
    exit(1);
  }
}


int main(int argc, char *argv[]) {
  
  /* Default argument values first. */
  struct arguments arguments;
  arguments.nloops = 100;
  //arguments.nthreads = 512;
  arguments.nthreads = 1024;
  //arguments.nantennas = 6;
  arguments.nantennas = 16;
  //arguments.nchannels = 2048;
  arguments.nchannels = 16384;
  //arguments.nsamples = 1<<24;
  arguments.nsamples = 1<<24;
  //arguments.bandwidth = 64e6;
  arguments.bandwidth = 2.048e9;
  arguments.verbose = 0;
  arguments.nbits = 2;
  //arguments.nbits = 4;
  arguments.complexdata = 0;
  arguments.jsonfile[0] = 0;
  int npolarisations = 2;
  hiprandGenerator_t gen;

  FILE *jsonvis = NULL;
  
  argp_parse(&argp, argc, argv, 0, 0, &arguments);


  // Initializing the tensor core correlator
  int NR_CHANNELS = 480;
  //int NR_POLARIZATIONS = 2;
  int NR_SAMPLES_PER_CHANNEL = 3072;
  int NR_RECEIVERS = 576;
  int NR_BASELINES = (NR_RECEIVERS) * ((NR_RECEIVERS) + 1) / 2;
  int NR_RECEIVERS_PER_BLOCK = 64;
  int NR_TIMES_PER_BLOCK = 128 / (NR_BITS);

  checkCudaCall(hipSetDevice(0)); // combine the CUDA runtime API and CUDA driver API
  checkCudaCall(hipFree(0));

  //tcc::Correlator tcc_correlator(NR_BITS, arguments.nantennas, arguments.nchannels, samples_per_channel, npolarisations, NR_RECEIVERS_PER_BLOCK);
  tcc::Correlator correlator(NR_BITS, NR_RECEIVERS, NR_CHANNELS, NR_SAMPLES_PER_CHANNEL, NR_POLARIZATIONS, NR_RECEIVERS_PER_BLOCK);

  hipStream_t stream_tcc;
  checkCudaCall(hipStreamCreate(&stream_tcc));

  //cu::Stream hostToDeviceStream, deviceToHostStream;
 
  //multi_array::extent<5> samplesExtent(multi_array::extents[NR_CHANNELS][NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK][NR_RECEIVERS][NR_POLARIZATIONS][NR_TIMES_PER_BLOCK]);
  //multi_array::extent<4> visibilitiesExtent(multi_array::extents[NR_CHANNELS][NR_BASELINES][NR_POLARIZATIONS][NR_POLARIZATIONS]);

  //cu::DeviceMemory deviceSamples_initial(sizeof(char2) * samplesExtent.size);

  char2 * deviceSamples_initial;
  hipMalloc((void **)&deviceSamples_initial, NR_CHANNELS * NR_SAMPLES_PER_CHANNEL * NR_RECEIVERS * NR_POLARIZATIONS * sizeof(char2));

  //cu::DeviceMemory deviceVisibilities(sizeof(Visibility) * visibilitiesExtent.size);
  //cu::HostMemory hostVisibilities(sizeof(Visibility) * visibilitiesExtent.size);
  //multi_array::array_ref<Visibility, 4> visibilitiesRef(* (Visibility *) hostVisibilities, visibilitiesExtent);

  std::complex<int32_t> * visibilities;
  checkCudaCall(hipMallocManaged(&visibilities, NR_CHANNELS * NR_BASELINES * NR_POLARIZATIONS * NR_POLARIZATIONS * sizeof(std::complex<int32_t>)));

  dim3 threads, blocks;
  threads.x = NR_TIMES_PER_BLOCK; threads.y = NR_POLARIZATIONS ; threads.z = 1;
  blocks.x = NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK; blocks.y = NR_CHANNELS ; blocks.z =  NR_RECEIVERS; 

 
  // Prepare a JSON file if necessary.
  if (strlen(arguments.jsonfile) > 0) {
    prepareJson(&jsonvis, arguments.jsonfile);
  }
  
  // Always discard the first trial.
  arguments.nloops += 1;

  // Calculate the samplegranularity
  int samplegranularity = 8 / (arguments.nbits * npolarisations);
  if (samplegranularity < 1)
  {
    samplegranularity = 1;
  }
  
  // Calculate the number of FFTs
  int fftsamples = arguments.nchannels * ((arguments.complexdata == 1) ? 1 : 2);
  int numffts = arguments.nsamples / fftsamples;
  printf("fftsamples = %d , numffts is %d\n", fftsamples, numffts);
  if (numffts % 8) {
    printf("Unable to proceed, numffts must be divisible by 8!\n");
    exit(0);
  }

  // Output our parameters.
  writeJsonValue(jsonvis, "int", 1, "nantennas", arguments.nantennas);
  writeJsonValue(jsonvis, "int", 0, "nsamples", arguments.nsamples);
  writeJsonValue(jsonvis, "int", 0, "nchannels", arguments.nchannels);
  writeJsonValue(jsonvis, "int", 0, "complexdata", arguments.complexdata);
  writeJsonValue(jsonvis, "int", 0, "samplegranularity", samplegranularity);
  writeJsonValue(jsonvis, "int", 0, "fftsamples", fftsamples);
  writeJsonValue(jsonvis, "int", 0, "numffts", numffts);
  
  printf("BENCHMARK PROGRAM STARTS\n\n");

  // Our collection of timers.
  struct timerCollection timers;
  timerInitialise(&timers);
  float timerResult;
  
  /*
   * This benchmarks unpacker kernels.
   */
  hipComplex **unpacked = new hipComplex*[arguments.nantennas * npolarisations];
  hipComplex **unpackedData, *unpackedData2;
  int8_t **packedData, **packedData8;
  int32_t *sampleShift;
  float implied_time;
  //dim3 FringeSetblocks, unpackBlocks;
  dim3 unpackBlocks;
  double *gpuDelays, **delays, *antfileoffsets;
  double lo, sampletime;
  float *rotationPhaseInfo, *fractionalSampleDelays;

  int i, j, unpackThreads, executionsperthread = 1, numkernelexecutions;
  int delayPhaseThreads;

  numkernelexecutions = fftsamples;
  if (numkernelexecutions <= arguments.nthreads) {
    unpackThreads = numkernelexecutions;
    executionsperthread = 1;
  } else {
    unpackThreads = arguments.nthreads;
    executionsperthread = numkernelexecutions / arguments.nthreads;
    if (numkernelexecutions % arguments.nthreads) {
      printf("Error: number of threads not divisible into number of kernel executions!\n");
      exit(0);
    }
  }

  unpackBlocks = dim3(executionsperthread, numffts);
  //FringeSetblocks = dim3(8, arguments.nantennas);

  numkernelexecutions = numffts;
  if (numkernelexecutions <= arguments.nthreads) {
    delayPhaseThreads = numkernelexecutions;
    executionsperthread = 1;
  } else {
    delayPhaseThreads = arguments.nthreads;
    executionsperthread = numkernelexecutions / arguments.nthreads;
    if (numkernelexecutions % arguments.nthreads) {
      printf("Error: number of threads not divisible into number of kernel executions!\n");
      exit(0);
    }
  }
  dim3 delayPhaseBlocks = dim3(executionsperthread, arguments.nantennas);
  
  // Allocate the memory.
  int max_delay_samples = 32768;
  int packedBytes = (arguments.nsamples + max_delay_samples) * 2 * npolarisations / 8;
  int packedBytes8 = packedBytes * 4;
  packedData = new int8_t*[arguments.nantennas];
  packedData8 = new int8_t*[arguments.nantennas];
  for (i = 0; i < arguments.nantennas; i++) {
    gpuErrchk(hipMalloc(&packedData[i], packedBytes));
    gpuErrchk(hipMalloc(&packedData8[i], packedBytes8));
  }
  for (i = 0; i < arguments.nantennas * npolarisations; i++) {
    gpuErrchk(hipMalloc(&unpacked[i], arguments.nsamples * sizeof(hipComplex)));
  }
  gpuErrchk(hipMalloc(&unpackedData, arguments.nantennas * npolarisations * sizeof(hipComplex*)));
  gpuErrchk(hipMemcpy(unpackedData, unpacked, arguments.nantennas * npolarisations * sizeof(hipComplex*), hipMemcpyHostToDevice));
  gpuErrchk(hipMalloc(&unpackedData2, arguments.nantennas * npolarisations * arguments.nsamples * sizeof(hipComplex)));

  /* Allocate memory for the sample shifts vector */
  gpuErrchk(hipMalloc(&sampleShift, arguments.nantennas * numffts * sizeof(int)));
  gpuErrchk(hipMemset(sampleShift, 0, arguments.nantennas * numffts * sizeof(int)));
  gpuErrchk(hipMalloc(&rotationPhaseInfo, arguments.nantennas * numffts * 2 * sizeof(float)));
  gpuErrchk(hipMalloc(&fractionalSampleDelays, arguments.nantennas * numffts * 2 * sizeof(float)));
  
  // Copy the delays to the GPU.
  gpuErrchk(hipMalloc(&gpuDelays, arguments.nantennas * 4 * sizeof(double)));
  delays = new double*[arguments.nantennas];
  antfileoffsets = new double[arguments.nantennas];
  srand(time(NULL));
  for (i = 0; i < arguments.nantennas; i++) {
    delays[i] = new double[3];
    for (j = 0; j < 3; j++) {
      delays[i][j] = (double)rand();
    }
    antfileoffsets[i] = (double)rand();
  }
  for (i = 0; i < arguments.nantennas; i++) {
    gpuErrchk(hipMemcpy(&(gpuDelays[i * 4]), delays[i], 3 * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(&(gpuDelays[i * 4 + 3]), &(antfileoffsets[i]), sizeof(double), hipMemcpyHostToDevice));
  }

  // Generate some random numbers, and some not so random.
  lo = (double)rand();
  sampletime = (arguments.complexdata == 1) ? (1.0 / arguments.bandwidth) : (1.0 / (2 * arguments.bandwidth));
  
  
  //unpackBlocks = arguments.nsamples / npolarisations / arguments.nthreads;
  printf("Each unpacking test will run with %d threads, %d x %d blocks\n", unpackThreads, unpackBlocks.x, unpackBlocks.y);
  printf("  nsamples = %d\n", arguments.nsamples);
  printf("  nantennas = %d\n", arguments.nantennas);
  
  // Generate some random data.
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
  for (i = 0; i < arguments.nantennas; i++) {
    hiprandGenerateUniform(gen, (float*)packedData[i], packedBytes * (sizeof(int8_t) / sizeof(float)));
    hiprandGenerateUniform(gen, (float*)packedData8[i], packedBytes8 * (sizeof(int8_t) / sizeof(float)));
  }
  hiprandDestroyGenerator(gen);
  hipGetLastError();


  dim3 threads_Gen, blocks_Gen;
  threads_Gen.x = 1024; threads_Gen.y = 1 ; threads_Gen.z = 1;
  blocks_Gen.x = (packedBytes8/1024); blocks_Gen.y = 1 ; blocks_Gen.z =  1; 

  for (i = 0; i < arguments.nantennas; i++) {
    generateRandomInt8<<<blocks_Gen, threads_Gen>>>(packedData8[i], packedBytes8);
  }


  timerAdd(&timers, "calculateDelaysAndPhases");
  timerAdd(&timers, "old_unpack2bit_2chan");
  timerAdd(&timers, "unpack2bit_2chan");
  timerAdd(&timers, "unpack2bit_2chan_fast");
  timerAdd(&timers, "unpack8bitcomplex_2chan");
  for (i = 0; i < arguments.nloops; i++) {
    if (arguments.verbose) {
      printf("\nLOOP %d\n", i);
    }

    // Run the delay calculator.
    if (arguments.verbose) {
      printf("  RUNNING DELAY KERNEL...");
      printf("   blocks = x: %d y: %d\n", delayPhaseBlocks.x, delayPhaseBlocks.y);
      printf("   threads = %d\n", delayPhaseThreads);
    }
    timerStart(&timers, "calculateDelaysAndPhases");
    calculateDelaysAndPhases<<<delayPhaseBlocks, delayPhaseThreads>>>(gpuDelays, lo, sampletime,
								      fftsamples,
								      arguments.nchannels,
								      samplegranularity,
								      rotationPhaseInfo,
								      sampleShift,
								      fractionalSampleDelays);
    gpuErrchk(hipPeekAtLastError());
    timerResult = timerEnd(&timers);
    if (arguments.verbose) {
      printf("  done in %8.3f ms.\n", timerResult);
    }
    
    // Now do the unpacking.
    if (arguments.verbose) {
      printf("  RUNNING KERNEL... ");
    }
    timerStart(&timers, "old_unpack2bit_2chan");
    for (j = 0; j < arguments.nantennas; j++) {
      old_unpack2bit_2chan<<<unpackBlocks, unpackThreads>>>(unpackedData, packedData[j], j);
      gpuErrchk(hipPeekAtLastError());
    }
    timerResult = timerEnd(&timers);
    if (arguments.verbose) {
      printf("  done in %8.3f ms.\n", timerResult);
    }

    if (arguments.verbose) {
      printf("  RUNNING KERNEL 2... ");
    }
    timerStart(&timers, "unpack2bit_2chan");
    for (j = 0; j < arguments.nantennas; j++) {
      unpack2bit_2chan<<<unpackBlocks, unpackThreads>>>(&unpackedData2[2*j*arguments.nsamples], packedData[j]);
      gpuErrchk(hipPeekAtLastError());
    }
    timerResult = timerEnd(&timers);
    if (arguments.verbose) {
      printf("  done in %8.3f ms.\n", timerResult);
    }

    if (arguments.verbose) {
      printf("  RUNNING KERNEL 3... ");
    }
    init_2bitLevels();
    timerStart(&timers, "unpack2bit_2chan_fast");
    for (j = 0; j < arguments.nantennas; j++) {
      unpack2bit_2chan_fast<<<unpackBlocks, unpackThreads>>>(&unpackedData2[2*j*arguments.nsamples], packedData[j], &(sampleShift[numffts*j]), fftsamples);
      gpuErrchk(hipPeekAtLastError());
    }
    timerResult = timerEnd(&timers);
    if (arguments.verbose) {
      printf("  done in %8.3f ms.\n", timerResult);
    }

    if (arguments.verbose) {
      printf("  RUNNING KERNEL 4... ");
    }
    init_2bitLevels();
    timerStart(&timers, "unpack8bitcomplex_2chan");
    for (j = 0; j < arguments.nantennas; j++) {
      unpack8bitcomplex_2chan<<<unpackBlocks, unpackThreads>>>(&unpackedData2[2*j*arguments.nsamples], packedData8[j], &(sampleShift[numffts*j]), fftsamples);
      gpuErrchk(hipPeekAtLastError());
    }
    timerResult = timerEnd(&timers);
    if (arguments.verbose) {
      printf("  done in %8.3f ms.\n", timerResult);
    }
  }
  implied_time = (float)arguments.nsamples;
  if (arguments.complexdata) {
    // Bandwidth is the same as the sampling rate.
    implied_time /= (float)arguments.bandwidth;
    // But the data is twice as big.
    implied_time /= 2;
  } else {
    implied_time /= 2 * (float)arguments.bandwidth;
  }
  timerPrintStatistics(&timers, "calculateDelaysAndPhases", implied_time, jsonvis);
  timerPrintStatistics(&timers, "old_unpack2bit_2chan", implied_time, jsonvis);
  timerPrintStatistics(&timers, "unpack2bit_2chan", implied_time, jsonvis);
  timerPrintStatistics(&timers, "unpack2bit_2chan_fast", implied_time, jsonvis);
  timerPrintStatistics(&timers, "unpack8bitcomplex_2chan", implied_time, jsonvis);

  // Free some memory.
  for (i = 0; i < arguments.nantennas; i++) {
    gpuErrchk(hipFree(packedData[i]));
    gpuErrchk(hipFree(packedData8[i]));
    gpuErrchk(hipFree(unpacked[i]));
  }
  gpuErrchk(hipFree(unpackedData));
  
  /*
   * This benchmarks the performance of the fringe rotator kernel.
   */
  hipComplex *unpackedFR;
  /* A suitable array has already been defined and populated. */
  unpackedFR = unpackedData2;
  float *rotVec;
  dim3 fringeBlocks;
  
  // Work out the block and thread numbers.
  fringeBlocks = dim3((arguments.nchannels / arguments.nthreads), numffts, arguments.nantennas);
  printf("\n\nEach fringe rotation test will run:\n");
  printf("  nsamples = %d\n", arguments.nsamples);
  printf("  nchannels = %d\n", arguments.nchannels);
  printf("  nffts = %d\n", numffts);
  
  //hipEventCreate(&start_test_fringerotate);
  //hipEventCreate(&end_test_fringerotate);

  /* Allocate memory for the rotation vector. */
  gpuErrchk(hipMalloc(&rotVec, arguments.nantennas * numffts * 2 * sizeof(float)));

  /* Fill it with random data. */
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
  hiprandGenerateUniform(gen, rotVec, arguments.nantennas * numffts * 2);
  hiprandDestroyGenerator(gen);
  hipGetLastError();

  timerAdd(&timers, "FringeRotate2");
  timerAdd(&timers, "FringeRotate");
  for (i = 0; i < arguments.nloops; i++) {
    
    timerStart(&timers, "FringeRotate2");
    FringeRotate2<<<fringeBlocks, arguments.nthreads>>>(unpackedFR, rotVec);
    gpuErrchk(hipPeekAtLastError());
    timerEnd(&timers);
    
    timerStart(&timers, "FringeRotate");
    FringeRotate<<<fringeBlocks, arguments.nthreads>>>(unpackedFR, rotVec);
    gpuErrchk(hipPeekAtLastError());
    timerEnd(&timers);
    
    //hipEventRecord(end_test_fringerotate, 0);
    //hipEventSynchronize(end_test_fringerotate);
    //hipEventElapsedTime(&(dtime_fringerotate[i]), start_test_fringerotate,
    //			 end_test_fringerotate);
    postLaunchCheck();
  }
  timerPrintStatistics(&timers, "FringeRotate", implied_time, jsonvis);
  timerPrintStatistics(&timers, "FringeRotate2", implied_time, jsonvis);

  /*
   * This benchmarks the performance of the FFT.
   */
  hipfftHandle plan;
  hipComplex *channelisedData;
  int nbaseline = arguments.nantennas * (arguments.nantennas - 1) / 2;
  int parallelAccum = (int)ceil(arguments.nthreads / arguments.nchannels + 1);
  int rc;
  while (parallelAccum && numffts % parallelAccum) parallelAccum--;
  if (parallelAccum == 0) {
    printf("Error: can not determine block size for the cross correlator!\n");
    exit(0);
  }

  printf("\n\nEach fringe rotation test will run:\n");
  printf("  parallelAccum = %d\n", parallelAccum);
  printf("  nbaselines = %d\n", nbaseline);
  
  /* Allocate the necessary arrays. */
  gpuErrchk(hipMalloc(&channelisedData, arguments.nantennas * npolarisations *
		       arguments.nsamples * sizeof(hipComplex)));
  rc = hipfftPlan1d(&plan, fftsamples, HIPFFT_C2C, 2 * arguments.nantennas * numffts);
  if (rc != HIPFFT_SUCCESS) {
    printf("FFT planning failed! %d\n", rc);
    exit(0);
  }

  timerAdd(&timers, "hipfftExecC2C");
  for (i = 0; i < arguments.nloops; i++) {

    timerStart(&timers, "hipfftExecC2C");
    if (hipfftExecC2C(plan, unpackedFR, channelisedData, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
      printf("FFT execution failed!\n");
      exit(0);
    }
    timerEnd(&timers);

  }
  hipfftDestroy(plan);
  timerPrintStatistics(&timers, "hipfftExecC2C", implied_time, jsonvis);

  // Free some memory.
  gpuErrchk(hipFree(rotVec));
  gpuErrchk(hipFree(unpackedData2));
  gpuErrchk(hipFree(sampleShift));
  gpuErrchk(hipFree(rotationPhaseInfo));
  gpuErrchk(hipFree(fractionalSampleDelays));
  gpuErrchk(hipFree(gpuDelays));
  
  /*
   * This benchmarks the performance of the cross-correlator and accumulator
   * combination.
   */
  //int corrThreads, blockchan, nchunk, ccblock_width = 128;
  int corrThreads, blockchan, nchunk, ccblock_width = 1024;
  hipComplex *baselineData;
  dim3 corrBlocks, accumBlocks, ccblock, ccblock2;
  
  gpuErrchk(hipMalloc(&baselineData, nbaseline * 4 * arguments.nchannels *
		       parallelAccum * sizeof(hipComplex)));

  /*
  if (arguments.nchannels <= 512) {
    corrThreads = arguments.nchannels;
    blockchan = 1;
  } else {
    corrThreads = 512;
    blockchan = arguments.nchannels / 512;
  }
  */

   if (arguments.nchannels <= 1024) {
    corrThreads = arguments.nchannels;
    blockchan = 1;
  } else {
    corrThreads = 1024;
    blockchan = arguments.nchannels / 1024;
  }


  corrBlocks = dim3(blockchan, parallelAccum);
  accumBlocks = dim3(blockchan, 4, nbaseline);
  ccblock = dim3((1 + (arguments.nchannels - 1) / ccblock_width),
		 arguments.nantennas - 1, arguments.nantennas - 1);
  ccblock2 = dim3((1 + (arguments.nchannels - 1) / ccblock_width),
		  (2 * arguments.nantennas -1), (2 * arguments.nantennas - 1));
  nchunk = numffts / parallelAccum;

  printf("\n\nEach cross correlation test will run:\n");
  printf("  parallelAccum = %d\n", parallelAccum);
  printf("  nbaselines = %d\n", nbaseline);
  printf("  corrThreads = %d\n", corrThreads);
  printf("  corrBlocks = x: %d , y: %d, z: %d\n", corrBlocks.x, corrBlocks.y, corrBlocks.z);
  printf("  accumBlocks = x: %d , y: %d, z: %d\n", accumBlocks.x, accumBlocks.y, accumBlocks.z);
  printf("  nchunk = %d\n", nchunk);

  printf("  ccblock_width = %d\n", ccblock_width);
  printf("  ccblock = x: %d , y: %d, z: %d\n", ccblock.x, ccblock.y, ccblock.z);
  printf("  ccblock2 = x: %d , y: %d, z: %d\n", ccblock2.x, ccblock2.y, ccblock2.z);

  timerAdd(&timers, "CrossCorr");
  timerAdd(&timers, "finaliseAccum");
  timerAdd(&timers, "CrossCorrAccumHoriz");
  timerAdd(&timers, "CCAH2");
  timerAdd(&timers, "CCAH3");
  timerAdd(&timers, "CCAH3-new");
  timerAdd(&timers, "CrossCorrShared"); 


  // Using the tensor core correlator
  timerAdd(&timers, "FormatConversion");
  timerAdd(&timers, "TCC");

  for (i = 0; i < 10; i++) {

    //timerStart(&timers, "FormatConversion");
    //GPU_Transpose_8bits<<<blocks_shared, threads_shared>>>(channelisedData, samples_char1, arguments.nchannels, samples_per_channel);
    //GPU_Reorder_after_transpose_8bits<<<blocks, threads>>>(samples_char1, samples_char2);
    //timerEnd(&timers);


    GPU_set_initial_value<<<blocks, threads>>>(deviceSamples_initial, NR_SAMPLES_PER_CHANNEL / 5 % NR_TIMES_PER_BLOCK, 0, 174, NR_SAMPLES_PER_CHANNEL / 5 / NR_TIMES_PER_BLOCK, NR_CHANNELS / 3, 418);

    timerStart(&timers, "TCC");

    correlator.launchAsync((hipStream_t) stream_tcc, (hipDeviceptr_t) visibilities, (hipDeviceptr_t) deviceSamples_initial);
    //correlator.launchAsync((hipStream_t) stream_tcc, deviceVisibilities, (hipDeviceptr_t) deviceSamples_initial);
    checkCudaCall(hipDeviceSynchronize());
    //deviceToHostStream.memcpyDtoHAsync(hostVisibilities, deviceVisibilities, visibilitiesRef.bytesize());
    timerEnd(&timers);
    std::cout << "real: " << (*(visibilities + 160*NR_BASELINES*NR_POLARIZATIONS*NR_POLARIZATIONS + 87745*NR_POLARIZATIONS*NR_POLARIZATIONS)).real() << std::endl;
    std::cout << "imag: " << (*(visibilities + 160*NR_BASELINES*NR_POLARIZATIONS*NR_POLARIZATIONS + 87745*NR_POLARIZATIONS*NR_POLARIZATIONS)).imag() << std::endl;
    //std::cout << "visbility: " << visibilitiesRef[160][87745][0][0] << std::endl;

  }

  timerPrintStatistics(&timers, "FormatConversion", implied_time, jsonvis);
  timerPrintStatistics(&timers, "TCC", implied_time, jsonvis);




  for (i = 0; i < arguments.nloops; i++) {

    timerStart(&timers, "CrossCorr");
    CrossCorr<<<corrBlocks, corrThreads>>>(channelisedData, baselineData,
					   arguments.nantennas, nchunk);
    gpuErrchk(hipPeekAtLastError());
    timerEnd(&timers);

    timerStart(&timers, "CrossCorrShared");
    CrossCorrShared<<<corrBlocks, corrThreads>>>(channelisedData, baselineData,
					   arguments.nantennas, nchunk);
    gpuErrchk(hipPeekAtLastError());
    timerEnd(&timers);

    
    timerStart(&timers, "finaliseAccum");
    finaliseAccum<<<accumBlocks, corrThreads>>>(baselineData, parallelAccum, nchunk);
    gpuErrchk(hipPeekAtLastError());
    timerEnd(&timers);

    timerStart(&timers, "CrossCorrAccumHoriz");
    CrossCorrAccumHoriz<<<ccblock, ccblock_width>>>(baselineData, channelisedData,
						    arguments.nantennas, numffts,
						    arguments.nchannels, fftsamples);
    gpuErrchk(hipPeekAtLastError());
    timerEnd(&timers);
    
    timerStart(&timers, "CCAH2");
    CCAH2<<<ccblock2, ccblock_width>>>(baselineData, channelisedData,
				      arguments.nantennas, numffts,
				      arguments.nchannels, fftsamples);
    gpuErrchk(hipPeekAtLastError());
    timerEnd(&timers);

    timerStart(&timers, "CCAH3");
    CCAH2<<<ccblock, ccblock_width>>>(baselineData, channelisedData,
				      arguments.nantennas, numffts,
				      arguments.nchannels, fftsamples);
    gpuErrchk(hipPeekAtLastError());
    timerEnd(&timers);

    timerStart(&timers, "CCAH3-new");
    CCAH3<<<ccblock, ccblock_width>>>(baselineData, channelisedData,
				      arguments.nantennas, numffts,
				      arguments.nchannels, fftsamples);
    gpuErrchk(hipPeekAtLastError());
    timerEnd(&timers);

  }

  timerPrintStatistics(&timers, "CrossCorr", implied_time, jsonvis);
  timerPrintStatistics(&timers, "finaliseAccum", implied_time, jsonvis);
  timerPrintStatistics(&timers, "CrossCorrAccumHoriz", implied_time, jsonvis);
  timerPrintStatistics(&timers, "CCAH2", implied_time, jsonvis);
  timerPrintStatistics(&timers, "CCAH3", implied_time, jsonvis);
  timerPrintStatistics(&timers, "CCAH3-new", implied_time, jsonvis); 

  timerPrintStatistics(&timers, "CrossCorrShared", implied_time, jsonvis);

  hipStream_t stream_DtoH;
  hipStreamCreate(&(stream_DtoH));
  hipComplex *baselineDatahost; 
  hipHostAlloc((void **)&baselineDatahost, (nbaseline * 4 * arguments.nchannels * parallelAccum * sizeof(hipComplex)), hipHostMallocDefault);
  hipMemcpyAsync(baselineDatahost, baselineData, (nbaseline * 4 * arguments.nchannels * parallelAccum * sizeof(hipComplex)), hipMemcpyDeviceToHost, stream_DtoH);
  hipDeviceSynchronize();

  //checkCudaCall(hipFree(visibilities));
  //checkCudaCall(hipFree(samples_char1));
  //checkCudaCall(hipFree(samples_char2));
  //free(samples_host);
  //checkCudaCall(hipFree(samples));
  //checkCudaCall(hipFree(visibilities));
  //checkCudaCall(hipFree(samples));
  checkCudaCall(hipFree(deviceSamples_initial));
  checkCudaCall(hipFree(visibilities));
  checkCudaCall(hipStreamDestroy(stream_tcc));
  hipStreamDestroy(stream_DtoH);
  hipHostFree(baselineDatahost);


  closeJson(jsonvis);
}


